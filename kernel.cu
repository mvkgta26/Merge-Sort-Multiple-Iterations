#include "hip/hip_runtime.h"
﻿#ifndef __HIPCC__
	#define __HIPCC__
#endif

#include <hip/hip_runtime.h>
#include ""
#include <hip/hip_runtime.h>
#include "hip/device_functions.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <time.h>
#include <windows.h>
#include < time.h >
#include <iostream>



//-------------------------------------------------------CPU TIMER LIBRARY-------------------------------------------------------

#if defined(_MSC_VER) || defined(_MSC_EXTENSIONS)
#define DELTA_EPOCH_IN_MICROSECS  116444736000000000Ui64 // CORRECT
#else
#define DELTA_EPOCH_IN_MICROSECS  116444736000000000ULL // CORRECT
#endif

struct timezone
{
	int  tz_minuteswest; /* minutes W of Greenwich */
	int  tz_dsttime;     /* type of dst correction */
};

// Definition of a gettimeofday function

int gettimeofday(struct timeval* tv, struct timezone* tz)
{
	// Define a structure to receive the current Windows filetime
	FILETIME ft;

	// Initialize the present time to 0 and the timezone to UTC
	unsigned __int64 tmpres = 0;
	static int tzflag = 0;

	if (NULL != tv)
	{
		GetSystemTimeAsFileTime(&ft);

		// The GetSystemTimeAsFileTime returns the number of 100 nanosecond 
		// intervals since Jan 1, 1601 in a structure. Copy the high bits to 
		// the 64 bit tmpres, shift it left by 32 then or in the low 32 bits.
		tmpres |= ft.dwHighDateTime;
		tmpres <<= 32;
		tmpres |= ft.dwLowDateTime;

		// Convert to microseconds by dividing by 10
		tmpres /= 10;

		// The Unix epoch starts on Jan 1 1970.  Need to subtract the difference 
		// in seconds from Jan 1 1601.
		tmpres -= DELTA_EPOCH_IN_MICROSECS;

		// Finally change microseconds to seconds and place in the seconds value. 
		// The modulus picks up the microseconds.
		tv->tv_sec = (long)(tmpres / 1000000UL);
		tv->tv_usec = (long)(tmpres % 1000000UL);
	}

	if (NULL != tz)
	{
		if (!tzflag)
		{
			_tzset();
			tzflag++;
		}

		// Adjust for the timezone west of Greenwich
		tz->tz_minuteswest = _timezone / 60;
		tz->tz_dsttime = _daylight;
	}

	return 0;
}

//--------------------------------------------------------GPU TIMER LIBRARY--------------------------------------------------------------------

struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float Elapsed()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};

//-----------------------------------------------------------------------------------------------------------------------------

using namespace std;



//Device function which bubble sorts a specific section of arr[]. Section [start:end]. Both inclusive. 
__device__ void bubblesort(int* arr, int start, int end)
{
	int n = end - start + 1; //Length of array section from start to end

	int i, k, flag, temp;
	for (k = 1; k < (n - 1) + 1; k++)
	{
		flag = 0;
		for (i = start; i < (end - k + 1); i++)
		{
			if (arr[i] > arr[i + 1])
			{
				temp = arr[i];    //
				arr[i] = arr[i + 1];  //  Swapping A[i+1] and A[i]
				arr[i + 1] = temp;  //
			}
		}
	}
}



//Entire block for the entire array. Each thread takes care of bubble-sorting an individual section of size : *section_length
__global__ void section_sort(int* nums, int section_size)   //(, int n)
{
	//The thread with thread index = idx will take care of nums[] from : [( section_size * idx ) to ( section_size * (idx + 1) - 1 )]
	//For example: idx = 1 and section_size = 20, then, thread with idx = 1 will take care of nums[ 20: 39 ]
	
	int idx = threadIdx.x;


	
	//Bubble sort nums[] from index  [ ( section_size * idx ) : ( section_size * (idx + 1) - 1 ) ]
	bubblesort(nums, section_size * idx, (section_size * (idx + 1) - 1));

}



//----------------------------------------------------------------------------------------------------------------------------

//Device Function: Takes a number target and searches the array arr[] ( an array of size n), and returns the index such that : nums[index] <= target
__device__ int bin_search(int* arr, int target, int n)        // n = size of arr2
{

	//Corner Cases : When the target is out of boundary of the range of values in the array
	if (target < arr[0])		return -1;
	if (target > arr[n - 1])    return n - 1;


	//f, l, mid
	int left = 0;
	int right = n;
	int mid = (left + right) / 2;

	while (left <= right)
	{
		int mid = (left + right) / 2;    //Calculate mid

		if (arr[mid] == target)
		{
			return mid - 1;   //Return index where nums[] == target
		}

		//All elements to right of mid are greater than target
		else if (arr[mid] > target)
		{
			//If nums[mid-1] < target < nums[mid]      ( Meaning target lies between nums[mid-1] and nums[mid] ==> (mid-1) is the required index)
			if (arr[mid - 1] < target)
			{
				return (mid - 1);
			}

			else			//Change the right border
			{
				right = mid - 1;
			}

		}

		//All elements to left of mid are lesser than target
		else if (arr[mid] < target)
		{
			//If nums[mid] < target < nums[mid+1]            ( Meaning target lies between nums[mid] and nums[mid+1] )
			if ((arr[mid + 1] > target))
			{
				return (mid);
			}


			else		//Change the left border
			{
				left = mid + 1;
			}

		}
	}

	return -1;
}


//Merges 2 sorted array, by using a GPU kernel call to parallely produce scatter addresses:
// Each Thread of block will parallely produce scatter addresses for its element. Block is divided into 2 sections. 
// Scatter address for All elements of both sections are parallelly produced.
// THe block is responsible for merging both of its sections
// Finally, the block in array is sorted according to the scatter addresses
	// *arr  = array pointer
	// *section_size = The length of the both 2 subarrays into which arr[] is split
	// *d_out_temp = Where array output is stored
__global__ void merge(int* arr, int section_length, int* d_out_temp)
{
	//int section_length = *section_size;
	int superset_length = section_length * 2;   //Block will be 2 * (size of 1 section). Because 2 sections are merged
	int idx = threadIdx.x;
	int b_idx = blockIdx.x;

	//Length of arr1[] and arr2[] are section size
	int len1 = section_length;
	int len2 = section_length;

	//-----Select *arr1 and *arr2 and *d_out_curr------------------------

	int* arr1 = arr + (b_idx * superset_length);
	int* arr2 = arr1 + (section_length);
	int* d_out_curr = d_out_temp + (b_idx * superset_length);   //Determine d_out_curr[], the output array for current merge 

	//Dynamically allocated shared memory array. 
	// scat_ad[] from index [0 to n1-1] is for arr1[].
	//scat_ad[] from index [n1 to n2-1] is for arr2[]

	 //Create a shared memory of size n1+n2 to accomodate the scatter-addresses corresonding to each element in arr1[] and arr2[] 
	extern __shared__ int scat_ad[];

	//--------------------------------These threads are responsible for arr1[]-------------------------------------------------------
	if (idx <= len1 - 1)
	{
		int idx1 = idx;     //Number of elements in arr1[] that are lesser than arr1[idx]. idx1 = index of current element in arr1[]

		int target = arr1[idx1];    //Target is current element in arr1[]

		//--------------Find idx2----------------------------------------Binary Search Part------------------------------
		int idx2 = bin_search(arr2, target, len2) + 1;    //Number of elements in arr2[] that are lesser than arr1[idx].....

		//Calculate and store the scatter address in array
		//scat_arr1[idx] = idx1 + idx2;     //If there are 2 elements before a number in output array, its index will be 2

		scat_ad[idx] = idx1 + idx2;    //Scatter address correspinding to arr1[idx] = idx1 + idx2 
	}


	//--------------------------------------These threads are responsible for arr2[]--------------------------------------------
	else if (idx >= len1)
	{
		//Number of elements in arr2[] that are lesser than arr2[idx]. 
		//idx1 = index of current element in arr2[]
		//(idx-len1) because threads with index n1 to n2-1 are responsible for arr2[] index [0: n2-1] 
		int idx1 = idx - len1;

		int target = arr2[idx1];    //Target is current element in arr1[] 

		//--------------Find idx2-----------------------------Binary Search Part---------------------------
		int idx2 = bin_search(arr1, target, len1) + 1;    //Number of elements in arr1[] that are lesser than arr2[idx].  +1 bcos we want appropriate position for current element

		//Calculate and store the scatter address in array
		//scat_arr1[idx] = idx1 + idx2;     //If there are 2 elements before a number in output array, its index will be 2
		scat_ad[idx] = idx1 + idx2;    //Scatter address corresponding to arr2[idx - len1] = idx1 + idx2 

	}

	__syncthreads();   //Barrier to ensure that all threads have finished writing scat_ad[].------------------Not necessary

	//-------------Store the output in respective position in d_out_temp[] using scatter address so that they are in sorted order-----------------------------------
	/*
	if (idx < len1)
	{
		d_out_curr[scat_ad[idx]] = arr1[idx];
	}
	else if (idx >= len1)
	{
		//d_out_curr[scat_ad[idx]] = arr2[idx - len1];
		d_out_curr[scat_ad[idx]] = arr1[idx];
	}
	*/
	d_out_curr[scat_ad[idx]] = arr1[idx];

	__syncthreads();
	//--------------------------------------Copy sorted elements back to array-----------------------------------------------------

	
	arr1[idx] = d_out_curr[idx];

	//printf( "%d ", arr1[idx] );
}



//Makes kernel call to merge 2 sorted array:
	//
void merge_sort()
{
	GpuTimer timer;

	//4 sections of 5 elements size
	//int h_arr[] = { 120,119,118,117,116,	115,114,113,112,111,	110,109,108,107,106,	105,104,103,102,101 };
	//int h_arr[] = { 596, 703, 277, 228, 548, 515, 213, 880, 391, 364, 224, 623, 845, 152, 454, 987, 854, 257, 402, 990, 996, 819, 756, 735, 460, 87, 693, 268, 92, 14, 860, 68, 996, 934, 478, 855, 209, 293, 171, 285 };
	int h_arr[40] = { 100,99,98,97,96,95,94,93,92,91,90,89,88,87,86,85,84,83,82,81,80,79,78,77,76,75,74,73,72,71,70,69,68,67,66,65,64,63,62,61 };
	//int h_arr[80] = { 100,99,98,97,96,95,94,93,92,91,90,89,88,87,86,85,84,83,82,81,80,79,78,77,76,75,74,73,72,71,70,69,68,67,66,65,64,63,62,61,60,59,58,57,56,55,54,53,52,51,50,49,48,47,46,45,44,43,42,41,40,39,38,37,36,35,34,33,32,31,30,29,28,27,26,25,24,23,22,21 };


	//int h_arr[64] = { 100,99,98,97,96,95,94,93,92,91,90,89,88,87,86,85,84,83,82,81,80,79,78,77,76,75,74,73,72,71,70,69,68,67,66,65,64,63,62,61,60,59,58,57,56,55,54,53,52,51,50,49,48,47,46,45,44,43,42,41,40,39,38,37 };


	int n = sizeof(h_arr) / sizeof(int);     //n = Total size of host array
	
	int div_num = 4;   //How many parts the array is initially split.
	int section_size = n/div_num;				// section_size = Size of each section after splitting arr[] into div_num parts (Stored in Host)


	//-----------------------------------Create input and output arrays in GPU---------------------------------------------
	int* d_arr, * d_out_temp;																// *d_out2;
	hipMalloc((void**)&d_arr, n * sizeof(int));
	hipMemcpy((void*)d_arr, (void*)h_arr, n * sizeof(int), hipMemcpyHostToDevice);   //d_arr[] is input array in device

	hipMalloc((void**)&d_out_temp, n * sizeof(int));			//d_out_temp[] is temporarily used to store sorted block elements
	


	timer.Start();

	//------------------------------Stage-1: KERNEL CALL: Bubble Sort Each Section of section_size elements------------------------------------

	section_sort <<<1, div_num>>> (d_arr, section_size);    //Call div_num threads: Each thread bubble-sorts a sub-section of n/div_num elements in the array.


	/*

	//---------------Stage-2 : KERNEL CALL: Perform 2 Parallel Merges on 2 Groups of 2 Sections (Each Section Of Size n/4)-----------------------------------------------
	
		//Make kernel call to 2 blocks of n/2 threads each. Each thread is responsible for 1 element of its block. 3rd parameter n/2 is for shared memory size
		//Imagine The entire arr[] is divided into 2 blocks of n/2 size each. Each block is divided into sections of section_size
	
	div_num = div_num / 2;				//Initially : Number of supersets will be Half of Total Number of Divisons (Here: 8/2 = 4)
	merge <<< div_num, n/div_num, n/div_num >>> ( d_arr, section_size, d_out_temp);
	 //Number of Threads = Size of Superset (Group of 2 sections)
	 //NUmber of blocks = Number of supersets

	//---------------------------Stage-3 : KERNEL CALL: Perform 1 Merge On 2 Sections (Each of Size n/2)-------------------------------------------------------------------
	
		//Make kernel call to 1 blocks of n threads . Each thread is responsible for 1 element of its block. 3rd parameter n is for shared memory size
		//Entire arr[] is 1 block. 

	div_num = div_num / 2;  //Number of supersets will be halved (Here : 4/2= 2)
	section_size = section_size * 2;    //Size of each section will double
	merge <<< div_num, n/div_num, n/div_num >>> ( d_arr, section_size, d_out_temp);      //Call kernel with INPUT: d_out_temp, and Output = d_out2[].... Section Length = 10 

	div_num = div_num / 2;  //Number of supersets will be halved (Here : 2/2 = 1)
	section_size = section_size * 2;    //Size of each section will double
	merge <<< div_num, n/div_num, n/div_num >>> (d_arr, section_size, d_out_temp);      //Call kernel with INPUT: d_out_temp, and Output = d_out2[].... Section Length = 10 


	*/

	//-----------------------------VERY IMPORTANT NOTE------------------------
	//NOTE: SUPERSET = GROUP OF 2 SECTIONS. WHEN WE MERGE A SUPERSET, WE MERGE THE 2 SECTIONS OF THE SUPERSET TO PRODUCE A SORTED SUPERSET
	
	//Initially, section_size = n / div_num
	int superset_num = div_num / 2; //Is the total number of supersets in the array, each of which are merged by a separate block. Initially, number of supersets = half of total number of divisions/sections in array
	while (superset_num >= 1)
	{

		/*
		//---------------Stage-suoerset_num : KERNEL CALL: Perform 2 Parallel Merges on 2 Groups of 2 Sections (Each Section Of Size n/4)-----------------------------------------------
		*/
		//Make kernel call to 2 blocks of n/2 threads each. Each thread is responsible for 1 element of its block. 3rd parameter n/2 is for shared memory size
		//Imagine The entire arr[] is divided into 2 blocks of n/2 size each. Each block is divided into sections of section_size

		//Number of Threads = Size of Superset 
		//Number of blocks = Number of supersets
		merge <<< superset_num, n/superset_num, n/superset_num >>> (d_arr, section_size, d_out_temp);
		
		//UPDATE : superset_num ( halved ) and section_size (doubled)
		superset_num = superset_num/2;
		section_size = section_size*2;

	}


	timer.Stop();

	double time_elapsed = timer.Elapsed();

		//---------------------------Copy Final Sorted Output From Device into a Host Array h_out[]------------------------------------------
	int* output_array = (int*)malloc( n * sizeof(int));
	
	hipMemcpy((void*)output_array, (void*)d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
	

	
	for (int i = 0; i < n; i++)
	{
		printf("%d ", output_array[i]);

		//if (i == 9) cout << endl;
	}

	printf("\n Time Elapsed : %g ms", time_elapsed);
		
}



void main()
{
	merge_sort();
}
